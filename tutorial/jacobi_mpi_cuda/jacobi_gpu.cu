#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016      The University of Tennessee and The University
 *                         of Tennessee Research Foundation.  All rights
 *                         reserved.
 * AUTHOR: George Bosilca
 */ 

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <mpi.h>
#include <stdio.h>
#include "header.h"

__global__ void __jacobi1( TYPE* nm, TYPE* om,
                           int nb, int mb )
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    int pos = 1 + i + (j+1) * (nb+2);

    nm[pos] = (om[pos - 1] +
               om[pos + 1] +
               om[pos - (nb+2)] +
               om[pos + (nb+2)]) / 4.0;
}

#define THREADS_PER_BLOCK_X 16
#define THREADS_PER_BLOCK_Y 16

#define CUDA_CHECK_ERROR( STR, ERROR, CODE )                            \
    {                                                                   \
        hipError_t __cuda_error = (hipError_t) (ERROR);               \
        if( hipSuccess != __cuda_error ) {                             \
            printf( "%s:%d %s %s\n", __FILE__, __LINE__,                \
                    (STR), hipGetErrorString(__cuda_error) );          \
            CODE;                                                       \
        }                                                               \
    }

extern "C" int preinit_jacobi_gpu(void)
{
    /* Interaction with the CUDA aware MPI library. In Open MPI CUDA
     * must be initialized before the MPI_Init in order to enable CUDA
     * support in the library.
     * In the case multiple GPUs are available per node and we have
     * multiple processes per node, let's distribute the processes
     * across all GPUs.
     */
    char* lrank = getenv("OMPI_COMM_WORLD_LOCAL_RANK");
    int local_rank, num_devices;
    if( NULL != lrank ) {
        local_rank = strtoul(lrank, NULL, 10);
    }
    hipGetDeviceCount(&num_devices);
    if( 0 == num_devices ) {
        printf("No CUDA devices on this node. Disable CUDA!\n");
    } else {
        hipSetDevice(local_rank % num_devices);
    }
    printf("Rank %d uses device %d\n", rank, local_rank % num_devices);
    
    return 0;
}

extern "C" int jacobi_gpu(TYPE* matrix, int N, int M, int P, MPI_Comm comm, TYPE epsilon)
{
    int NB, MB, Q, iter = 0;
    int rank, size, ew_rank, ew_size, ns_rank, ns_size;
    TYPE *d_om, *d_nm, *tmpm, *send_east, *send_west, *recv_east, *recv_west, diff_norm;
    TYPE *send_north, *send_south, *recv_north, *recv_south;
    hipError_t cudaStatus;
    MPI_Comm ns, ew;
    MPI_Request req[8] = {MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL,
                          MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL};

    MPI_Comm_rank(comm, &rank);
    MPI_Comm_size(comm, &size);
    Q = 1 + (size - 1) / P;
    NB = N / P;
    MB = M / Q;

    dim3 dimBlock(THREADS_PER_BLOCK_X,THREADS_PER_BLOCK_Y);
    dim3 dimGrid(NB/dimBlock.x,MB/dimBlock.y);

    cudaStatus = hipMalloc((void**)&d_om, sizeof(TYPE) * (NB+2) * (MB+2));
    CUDA_CHECK_ERROR( "hipMalloc", cudaStatus, { return -1; } );
    cudaStatus = hipMalloc((void**)&d_nm, sizeof(TYPE) * (NB+2) * (MB+2));
    CUDA_CHECK_ERROR( "hipMalloc", cudaStatus, { return -1; } );
    cudaStatus = hipMemcpy(d_om, matrix, sizeof(TYPE) * (NB+2) * (MB+2), hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR( "hipMemcpy", cudaStatus, { return -1; } );

    cudaStatus = hipHostMalloc((void**)&send_east, sizeof(TYPE) * MB);
    CUDA_CHECK_ERROR( "hipMalloc", cudaStatus, { return -1; } );
    cudaStatus = hipHostMalloc((void**)&send_west, sizeof(TYPE) * MB);
    CUDA_CHECK_ERROR( "hipMalloc", cudaStatus, { return -1; } );
    cudaStatus = hipHostMalloc((void**)&recv_east, sizeof(TYPE) * MB);
    CUDA_CHECK_ERROR( "hipMalloc", cudaStatus, { return -1; } );
    cudaStatus = hipHostMalloc((void**)&recv_west, sizeof(TYPE) * MB);
    CUDA_CHECK_ERROR( "hipMalloc", cudaStatus, { return -1; } );
    cudaStatus = hipHostMalloc((void**)&send_north, sizeof(TYPE) * NB);
    CUDA_CHECK_ERROR( "hipMalloc", cudaStatus, { return -1; } );
    cudaStatus = hipHostMalloc((void**)&send_south, sizeof(TYPE) * NB);
    CUDA_CHECK_ERROR( "hipMalloc", cudaStatus, { return -1; } );
    cudaStatus = hipHostMalloc((void**)&recv_north, sizeof(TYPE) * NB);
    CUDA_CHECK_ERROR( "hipMalloc", cudaStatus, { return -1; } );
    cudaStatus = hipHostMalloc((void**)&recv_south, sizeof(TYPE) * NB);
    CUDA_CHECK_ERROR( "hipMalloc", cudaStatus, { return -1; } );

    /* create the north-south and east-west communicator */
    MPI_Comm_split(comm, rank % P, rank, &ns);
    MPI_Comm_rank(ns, &ns_rank);
    MPI_Comm_size(ns, &ns_size);
    MPI_Comm_split(comm, rank / P, rank, &ew);
    MPI_Comm_rank(ew, &ew_rank);
    MPI_Comm_size(ew, &ew_size);

    printf("Rank %d/%d in MPI_COMM_WORLD is EW (rank %d, size %d) and NS (rank %d, size %d) [P=%d]\n",
           rank, size, ew_rank, ew_size, ns_rank, ns_size, P);

    /* Coordination events */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    do {
        /* Bring the data on the CPU */
        if( 0 != ns_rank ) {
            cudaStatus = hipMemcpyAsync(send_north, SEND_NORTH(d_om), sizeof(TYPE) * NB, hipMemcpyDeviceToHost, 0);
            CUDA_CHECK_ERROR( "hipMemcpyAsync(send, north)", cudaStatus, { return -1; } );
        }
        if( (ns_size-1) != ns_rank ) {
            cudaStatus = hipMemcpyAsync(send_south, SEND_SOUTH(d_om), sizeof(TYPE) * NB, hipMemcpyDeviceToHost, 0);
            CUDA_CHECK_ERROR( "hipMemcpyAsync(send, south)", cudaStatus, { return -1; } );
        }
        if( 0 != ew_rank ) {
            cudaStatus = hipMemcpy2DAsync(send_east, sizeof(TYPE), d_om + NB + 2 + 1, sizeof(TYPE) * (NB+2),
                                           sizeof(TYPE) * 1, NB, hipMemcpyDeviceToHost, 0);
            CUDA_CHECK_ERROR( "hipMemcpyAsync(send, east)", cudaStatus, { return -1; } );
        }
        if( (ew_size-1) != ew_rank) {
            cudaStatus = hipMemcpy2DAsync(send_west, sizeof(TYPE), d_om + NB + 2 + NB, sizeof(TYPE) * (NB+2),
                                           sizeof(TYPE) * 1, NB, hipMemcpyDeviceToHost, 0);
            CUDA_CHECK_ERROR( "hipMemcpyAsync(send, west)", cudaStatus, { return -1; } );
        }
        hipEventRecord(start, 0);

        /* post receives from the neighbors */
        if( 0 != ns_rank )
            MPI_Irecv( recv_north, NB, MPI_TYPE, ns_rank - 1, 0, ns, &req[0]);
        if( (ns_size-1) != ns_rank )
            MPI_Irecv( recv_south, NB, MPI_TYPE, ns_rank + 1, 0, ns, &req[1]);
        if( 0 != ew_rank )
            MPI_Irecv( recv_west,  MB, MPI_TYPE, ew_rank - 1, 0, ew, &req[3]);
        if( (ew_size-1) != ew_rank )
            MPI_Irecv( recv_east,  MB, MPI_TYPE, ew_rank + 1, 0, ew, &req[2]);

        cudaStatus = hipEventSynchronize(start);
        CUDA_CHECK_ERROR( "hipEventSynchronize", cudaStatus, { return -1; } );

        /* post the sends */
        if( 0 != ns_rank )
            MPI_Isend( send_north, NB, MPI_TYPE, ns_rank - 1, 0, ns, &req[4]);
        if( (ns_size-1) != ns_rank )
            MPI_Isend( send_south, NB, MPI_TYPE, ns_rank + 1, 0, ns, &req[5]);
        if( 0 != ew_rank )
            MPI_Isend( send_west,  MB, MPI_TYPE, ew_rank - 1, 0, ew, &req[7]);
        if( (ew_size-1) != ew_rank)
            MPI_Isend( send_east,  MB, MPI_TYPE, ew_rank + 1, 0, ew, &req[6]);
        /* wait until they all complete */
        MPI_Waitall(8, req, MPI_STATUSES_IGNORE);

        /* unpack the newly received data */
        if( 0 != ns_rank ) {
            cudaStatus = hipMemcpyAsync(RECV_NORTH(d_om), recv_north, sizeof(TYPE) * NB, hipMemcpyHostToDevice, 0);
            CUDA_CHECK_ERROR( "hipMemcpyAsync", cudaStatus, { return -1; } );
        }
        if( (ns_size-1) != ns_rank ) {
            cudaStatus = hipMemcpyAsync(RECV_SOUTH(d_om), recv_south, sizeof(TYPE) * NB, hipMemcpyHostToDevice, 0);
            CUDA_CHECK_ERROR( "hipMemcpyAsync", cudaStatus, { return -1; } );
        }
        if( 0 != ew_rank ) {
            cudaStatus = hipMemcpy2DAsync(d_om + NB + 2, sizeof(TYPE) * (NB+2), recv_east, sizeof(TYPE),
                                           sizeof(TYPE) * 1, NB, hipMemcpyHostToDevice, 0);
            CUDA_CHECK_ERROR( "hipMemcpyAsync", cudaStatus, { return -1; } );
        }
        if( (ew_size-1) != ew_rank) {
            cudaStatus = hipMemcpy2DAsync(d_om + NB + 2 + NB + 1, sizeof(TYPE) * (NB+2), recv_west, sizeof(TYPE),
                                           sizeof(TYPE) * 1, NB, hipMemcpyHostToDevice, 0);
            CUDA_CHECK_ERROR( "hipMemcpyAsync", cudaStatus, { return -1; } );
        }
        hipEventRecord(start, 0);

        /**
         * dimGrid blocks each one of dimBlock dimensions.
         */
        __jacobi1<<<dimGrid, dimBlock>>>(d_nm, d_om, NB, MB);

        hipDeviceSynchronize();
        cudaStatus = hipGetLastError();
        CUDA_CHECK_ERROR( "__jacobi1 kernel", cudaStatus, { return -1; } );

        cudaStatus = hipEventRecord(stop);
        CUDA_CHECK_ERROR( "hipEventRecord", cudaStatus, { return -1; } );
        cudaStatus = hipEventSynchronize(stop);
        CUDA_CHECK_ERROR( "hipEventSynchronize", cudaStatus, { return -1; } );

        diff_norm = epsilon + 1.0;  /* don't update epsilon */
        tmpm = d_om; d_om = d_nm; d_nm = tmpm;  /* swap the 2 matrices */
        iter++;
        MPI_Allreduce(MPI_IN_PLACE, &diff_norm, 1, MPI_TYPE, MPI_SUM,
                      MPI_COMM_WORLD);
        if(0 == rank) {
            printf("Iteration %d norm %f\n", iter, diff_norm);
        }
    } while((iter < MAX_ITER) && (sqrt(diff_norm) > epsilon));

    /* Update the matrix */
    cudaStatus = hipMemcpy(matrix, d_om, sizeof(TYPE) * (NB+2) * (MB+2), hipMemcpyDeviceToHost);
    CUDA_CHECK_ERROR( "hipMemcpy", cudaStatus, { return -1; } );

    hipFree(d_om);
    hipFree(d_nm);
    hipHostFree(send_west); hipHostFree(send_east);
    hipHostFree(send_north); hipHostFree(send_south);
    hipHostFree(recv_west); hipHostFree(recv_east);
    hipHostFree(recv_north); hipHostFree(recv_south);

    MPI_Comm_free(&ns);
    MPI_Comm_free(&ew);

    return iter;
}
